#include <algorithm>
#include <assert.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

int N = 1024;		// length of vector A
float* d_A = NULL;	// Pointer to vector A in device memory

double time_memcpy = 0;
double time_compute = 0;	// Timers

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on CPU: Policy 1
/////////////////////////////////////////////////////////////////////////
void inc_cpu(float* A, int n, int inc_count) {
	for (int i=0; i<n; i++) {
		for (int j=0; j<inc_count; j++) {
			A[i] += 1;
		}
	}	
}

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on GPU: Policy 2
/////////////////////////////////////////////////////////////////////////
__global__ void inc_gpu(float* A, int n, int inc_count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (int j=0; j<inc_count; j++) {
		A[i] += 1;
	}
}

/////////////////////////////////////////////////////////////////////////
// Macro to check that CUDA API calls are successful
/////////////////////////////////////////////////////////////////////////
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/////////////////////////////////////////////////////////////////////////
// Allocate device memory and copy from host to device memory
/////////////////////////////////////////////////////////////////////////
void copy_host_to_device(float* A, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	hipMalloc((void**) &d_A, sizeof(float) * n * n);
	assert(d_A);
	hipMemcpy(d_A, A, sizeof(float) * n, hipMemcpyHostToDevice);

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Copy result from device to host memory and free device memory
/////////////////////////////////////////////////////////////////////////
void copy_device_to_host(float* A, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	// TODO:
	// 1. Copy result array d_C to C using cudaMemcpy.
	// 2. Free memory allocated for d_A, d_B, d_C using cudaFree.

	hipMemcpy(A, d_A, sizeof(float) * n, hipMemcpyDeviceToHost);
	hipFree(d_A);

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	if(argc != 5) {printf("Usage: mat_mul_gpu <array size> <block size> <policy (1|2|3)> <(debug|nodebug)>\n") ; exit(2) ;}
	int inc_count = atoi(argv[1]) ;		// number of times to increment each element
	int block_size = atoi(argv[2]) ;	// size of thread block
	int policy = atoi(argv[3]) ;		// 1 or 2
	bool debug = (!strcmp(argv[4], "debug")) ? true : false;

	// allocate host memory for vector A
	float* h_A = (float*)malloc(sizeof(float) * N);

	// initialize host memory with zeros
	for (int i = 0; i < N; ++i){
		h_A[i] = 0;
	}

	//call the relevant policy 
	if(policy==1){ // CPU baseline
		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		inc_cpu(h_A, N, inc_count);

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;
	}
	else if(policy==2){ // GPU
		copy_host_to_device(h_A, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

                dim3 dimBlock(block_size);
                dim3 dimGrid(N / dimBlock.x);
		inc_gpu <<<dimGrid, dimBlock>>> (d_A, N, inc_count);

		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_A, N);
	}

	if (debug) {
		// print selected results (for checking correctness)
		for (int i=0; i<N; i+=N/10) { printf(" %10.6f",h_A[i]); }
		printf("\n");
	}
	if(!debug) {
		printf("Memcpy Time: %f seconds\n", time_memcpy);
		printf("Compute Time: %f seconds\n", time_compute);
	}

	// clean up memory allocated  
	free(h_A);

	hipDeviceReset();
}
