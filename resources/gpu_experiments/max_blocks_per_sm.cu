#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return (double) tp.tv_sec + (double)tp.tv_usec*1e-6;
}

__device__ void sleep(float t, clock_t clock_rate) {    
    clock_t t0 = clock64();
    clock_t t1 = t0;
    while ((t1 - t0)/(clock_rate*1000.0f) < t)
        t1 = clock64();
}

__global__ void mykernel(clock_t clock_rate) {
    sleep(1.0, clock_rate);    
}

int main(int argc, char* argv[]) {
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0); 
    int mp = prop.multiProcessorCount;
    clock_t clock_rate = prop.clockRate;

    int max_num_blocks = atoi(argv[1]);
    
    for (int num_blocks = 1; num_blocks <= max_num_blocks; num_blocks++) {
	dim3 block(1);
	dim3 grid(num_blocks);  /* N blocks */

	double start = cpuSecond();
	mykernel<<<grid,block>>>(clock_rate);
	hipDeviceSynchronize();
	double etime = cpuSecond() - start;

	printf("blocks      %10d\n",num_blocks);
	printf("SMs         %10d\n",mp);
	printf("blocks/SM   %10.2f\n",num_blocks/((double)mp));
	printf("time        %10.2f\n",etime);
        printf("\n");
    }

    hipDeviceReset();
}
