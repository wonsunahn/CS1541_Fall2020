#include <algorithm>
#include <assert.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

float* d_A = NULL;	// Pointer to matrix A in device memory
float* d_B = NULL;	// Pointer to matrix B in device memory
float* d_C = NULL;	// Pointer to result matrix C in device memory

double time_memcpy = 0;
double time_compute = 0;	// Timers

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on CPU: Policy 1
/////////////////////////////////////////////////////////////////////////
void mm_cpu(float* C, const float* A, const float* B, int n) {
	for (int i=0; i<n; i++) {
		for (int j=0; j<n; j++) {
                        C[i*n + j] = 0;
			for (int k=0; k<n; k++) {
				C[i*n + j] += A[i*n + k] * B[k*n + j];
			}
		}
	}
}

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on GPU: Policy 2
/////////////////////////////////////////////////////////////////////////
__global__ void mm_gpu(float* C, float* A, float* B, int n)
{
	// TODO:
	// Implement the naive GPU matrix multiplication shown on the lecture slides.
	// Remove the printf below when you don't need it.  It's just for demonstration.
	printf("Grid(%d, %d) Block (%d, %d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on GPU using shared memory: Policy 3
/////////////////////////////////////////////////////////////////////////
__global__ void mm_gpu_shared(float* C, float* A, float* B, int n)
{
	int block_size = blockDim.x;
	// Shared memory used to store a tile in arrays A and B respectively.
	// We needed to allocate dynamic shared memory because the block size is unknown ahead of time.
        // This looks strange but if you read the below NVIDIA blog entry, it will make sense:
	// https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/
        // This is equivalent to the below static shared memory declarations (but it won't work because of bock_size).
	// __shared__ float As[block_size][block_size];
	// __shared__ float Bs[block_size][block_size];
	extern __shared__ float As[];
	float *Bs = &As[block_size * block_size];

	// TODO:
	// Implement GPU matrix multiplication using tiling and the above allocated shared memory as described in:
	// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#shared-memory-in-matrix-multiplication-c-ab
}

/////////////////////////////////////////////////////////////////////////
// Macro to check that CUDA API calls are successful
/////////////////////////////////////////////////////////////////////////
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/////////////////////////////////////////////////////////////////////////
// Allocate device memory and copy from host to device memory
/////////////////////////////////////////////////////////////////////////
void copy_host_to_device(float* A, float* B, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	// TODO:
	// 1. Allocate GPU memory for d_A, d_B, d_C using cudaMalloc.
	// 2. Copy input arrays A, B to d_A, d_B using cudaMemcpy.

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Copy result from device to host memory and free device memory
/////////////////////////////////////////////////////////////////////////
void copy_device_to_host(float* C, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	// TODO:
	// 1. Copy result array d_C to C using cudaMemcpy.
	// 2. Free memory allocated for d_A, d_B, d_C using cudaFree.

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	if(argc != 5) {printf("Usage: mat_mul_gpu <array size> <block size> <policy (1|2|3)> <(debug|nodebug)>\n") ; exit(2) ;}
	int N = atoi(argv[1]) ;				// array size
	int block_size = atoi(argv[2]) ;	// size of thread block
	int policy = atoi(argv[3]) ;		// 1, 2 or 3
	bool debug = (!strcmp(argv[4], "debug")) ? true : false;

	//printf("\n Array size = %d , block size = %d and policy = %d\n ", N, block_size, policy);

	// set seed for rand()
	srand(2020);

	// allocate host memory for matrix A and vector x
	int matrix_size = N * N;
	float* h_A = (float*)malloc(sizeof(float) * matrix_size);
	float* h_B = (float*)malloc(sizeof(float) * matrix_size);
	float* h_C = (float*)malloc(sizeof(float) * matrix_size);

	// randomly initialize host memory
	for (int i = 0; i < matrix_size; ++i){
		h_A[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < matrix_size; i++){
		h_B[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < matrix_size; i++){
		h_C[i] = 0.0;
	}

	//call the relevant policy 
	if(policy==1){ //CPU baseline
		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		mm_cpu(h_C, h_A, h_B, N);

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;
	}
	else if(policy==2){ //CUDA MV multiplication
		copy_host_to_device(h_A, h_B, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		// TODO: Call the kernel
		// Call mm_gpu <<< >>> ( ) with the appropriate grid and thread block layouts.

		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_C, N);
	}
	else if(policy==3){ //CUDA MV multiplication with shared memory
		copy_host_to_device(h_A, h_B, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

                dim3 dimBlock(block_size, block_size);
                dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
		// Additional 3rd parameter "sizeof(float) * block_size * block_size * 2" is used to specify how much shared memory to allocate for a thread block.
		// It's needed because we want to dynamically allocate shared memory according to the given thread block size.
		mm_gpu_shared <<<dimGrid, dimBlock, sizeof(float) * block_size * block_size * 2>>> (d_C, d_A, d_B, N);

		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_C, N);
	}

	if (debug) {
		// print selected results (for checking correctness)
		for (int i=0; i<matrix_size; i+=matrix_size/10) { printf(" %10.6f",h_C[i]); }
		printf("\n");
	}
	if(!debug) {
		printf("Memcpy Time: %f seconds\n", time_memcpy);
		printf("Compute Time: %f seconds\n", time_compute);
	}

	// clean up memory allocated  
	free(h_A);
	free(h_B);
	free(h_C);

	hipDeviceReset();
}
